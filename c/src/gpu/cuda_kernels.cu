#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <npp.h>
#include "optimizer_gpu.h"

// Texture objects for efficient image access
texture<float4, hipTextureType2D, hipReadModeElementType> texInput;
texture<float4, hipTextureType2D, hipReadModeElementType> texWatermark;

// Constants
#define BLOCK_SIZE 16
#define MAX_THREADS_PER_BLOCK 1024

// Helper functions
__device__ float4 apply_color_correction(float4 color, float contrast, float brightness, float saturation) {
    // Convert to HSV
    float minVal = fminf(fminf(color.x, color.y), color.z);
    float maxVal = fmaxf(fmaxf(color.x, color.y), color.z);
    float delta = maxVal - minVal;
    
    float h = 0;
    float s = maxVal == 0 ? 0 : delta / maxVal;
    float v = maxVal;
    
    if (delta > 0) {
        if (maxVal == color.x) {
            h = (color.y - color.z) / delta + (color.y < color.z ? 6.0f : 0.0f);
        } else if (maxVal == color.y) {
            h = (color.z - color.x) / delta + 2.0f;
        } else {
            h = (color.x - color.y) / delta + 4.0f;
        }
        h /= 6.0f;
    }
    
    // Apply adjustments
    s = fminf(1.0f, s * saturation);
    v = fminf(1.0f, v * brightness);
    v = (v - 0.5f) * contrast + 0.5f;
    v = fmaxf(0.0f, fminf(1.0f, v));
    
    // Convert back to RGB
    float c = v * s;
    float x = c * (1 - fabsf(fmodf(h * 6.0f, 2.0f) - 1));
    float m = v - c;
    
    float4 result;
    if (h < 1.0f/6.0f) {
        result = make_float4(c + m, x + m, m, color.w);
    } else if (h < 2.0f/6.0f) {
        result = make_float4(x + m, c + m, m, color.w);
    } else if (h < 3.0f/6.0f) {
        result = make_float4(m, c + m, x + m, color.w);
    } else if (h < 4.0f/6.0f) {
        result = make_float4(m, x + m, c + m, color.w);
    } else if (h < 5.0f/6.0f) {
        result = make_float4(x + m, m, c + m, color.w);
    } else {
        result = make_float4(c + m, m, x + m, color.w);
    }
    
    return result;
}

__device__ float4 apply_sharpening(float4 center, float4 top, float4 bottom, float4 left, float4 right, float strength) {
    float4 laplacian = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    laplacian.x = 4 * center.x - top.x - bottom.x - left.x - right.x;
    laplacian.y = 4 * center.y - top.y - bottom.y - left.y - right.y;
    laplacian.z = 4 * center.z - top.z - bottom.z - left.z - right.z;
    
    float4 result;
    result.x = center.x + strength * laplacian.x;
    result.y = center.y + strength * laplacian.y;
    result.z = center.z + strength * laplacian.z;
    result.w = center.w;
    
    return result;
}

// Kernel implementations
extern "C" {

__global__ void resizeKernel(
    float4* output,
    int outputWidth,
    int outputHeight,
    float scaleX,
    float scaleY
) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= outputWidth || y >= outputHeight)
        return;
        
    float u = (x + 0.5f) * scaleX;
    float v = (y + 0.5f) * scaleY;
    
    float4 pixel = tex2D(texInput, u, v);
    output[y * outputWidth + x] = pixel;
}

__global__ void colorCorrectKernel(
    float4* output,
    int width,
    int height,
    float contrast,
    float brightness,
    float saturation
) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height)
        return;
        
    float4 pixel = tex2D(texInput, x + 0.5f, y + 0.5f);
    pixel = apply_color_correction(pixel, contrast, brightness, saturation);
    output[y * width + x] = pixel;
}

__global__ void sharpenKernel(
    float4* output,
    int width,
    int height,
    float strength
) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height)
        return;
        
    float4 center = tex2D(texInput, x + 0.5f, y + 0.5f);
    float4 top = tex2D(texInput, x + 0.5f, y - 0.5f);
    float4 bottom = tex2D(texInput, x + 0.5f, y + 1.5f);
    float4 left = tex2D(texInput, x - 0.5f, y + 0.5f);
    float4 right = tex2D(texInput, x + 1.5f, y + 0.5f);
    
    output[y * width + x] = apply_sharpening(center, top, bottom, left, right, strength);
}

__global__ void watermarkKernel(
    float4* output,
    int width,
    int height,
    int watermarkWidth,
    int watermarkHeight,
    int offsetX,
    int offsetY,
    float opacity
) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height)
        return;
        
    float4 pixel = tex2D(texInput, x + 0.5f, y + 0.5f);
    
    // Check if pixel is in watermark area
    if (x >= offsetX && x < offsetX + watermarkWidth &&
        y >= offsetY && y < offsetY + watermarkHeight) {
        float watermarkU = (float)(x - offsetX) / watermarkWidth;
        float watermarkV = (float)(y - offsetY) / watermarkHeight;
        float4 watermark = tex2D(texWatermark, watermarkU, watermarkV);
        
        // Alpha blending
        float alpha = watermark.w * opacity;
        pixel.x = (1 - alpha) * pixel.x + alpha * watermark.x;
        pixel.y = (1 - alpha) * pixel.y + alpha * watermark.y;
        pixel.z = (1 - alpha) * pixel.z + alpha * watermark.z;
    }
    
    output[y * width + x] = pixel;
}

} // extern "C"

// Kernel launch helpers
hipError_t launchResizeKernel(
    float4* output,
    int outputWidth,
    int outputHeight,
    float scaleX,
    float scaleY
) {
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(
        (outputWidth + block.x - 1) / block.x,
        (outputHeight + block.y - 1) / block.y
    );
    
    resizeKernel<<<grid, block>>>(
        output,
        outputWidth,
        outputHeight,
        scaleX,
        scaleY
    );
    
    return hipGetLastError();
}

hipError_t launchColorCorrectKernel(
    float4* output,
    int width,
    int height,
    float contrast,
    float brightness,
    float saturation
) {
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(
        (width + block.x - 1) / block.x,
        (height + block.y - 1) / block.y
    );
    
    colorCorrectKernel<<<grid, block>>>(
        output,
        width,
        height,
        contrast,
        brightness,
        saturation
    );
    
    return hipGetLastError();
}

hipError_t launchSharpenKernel(
    float4* output,
    int width,
    int height,
    float strength
) {
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(
        (width + block.x - 1) / block.x,
        (height + block.y - 1) / block.y
    );
    
    sharpenKernel<<<grid, block>>>(
        output,
        width,
        height,
        strength
    );
    
    return hipGetLastError();
}

hipError_t launchWatermarkKernel(
    float4* output,
    int width,
    int height,
    int watermarkWidth,
    int watermarkHeight,
    int offsetX,
    int offsetY,
    float opacity
) {
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(
        (width + block.x - 1) / block.x,
        (height + block.y - 1) / block.y
    );
    
    watermarkKernel<<<grid, block>>>(
        output,
        width,
        height,
        watermarkWidth,
        watermarkHeight,
        offsetX,
        offsetY,
        opacity
    );
    
    return hipGetLastError();
} 